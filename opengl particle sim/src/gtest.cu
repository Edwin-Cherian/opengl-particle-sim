#include "hip/hip_runtime.h"
#include "gtest.h"

__global__ void g_add(int n, float* x, float* y)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//int stride = blockDim.x * gridDim.x;
	//for (int i = index; i < 1 ; i += stride)
	y[idx] += x[idx];
}

__global__ void g_setup(float* x, float* y)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	x[idx] = 1.0f;
	y[idx] = 2.0f;
}

int g_main()
{
	int n = 1 << 20;
	float* x;
	float* y;
	hipMallocManaged(&x, n * sizeof(float));
	hipMallocManaged(&y, n * sizeof(float));

	int tpb = 256;
	int bpg = (n + tpb - 1) / tpb;

	g_setup<<<bpg, tpb>>>(x, y);

	auto start = std::chrono::high_resolution_clock::now();
	g_add<<<bpg, tpb>>>(n, x, y);
	hipDeviceSynchronize();
	auto stop = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
	std::cout << duration.count() << std::endl;

	float maxError = 0.0f;
	for (int i = 0; i < n; i++)
	{
		maxError = fmax(maxError, fabs(y[i] - 3.0f));
	}
	std::cout << "Max Error : " << maxError << std::endl;

	hipFree(x);
	hipFree(y);
	return 690;
}